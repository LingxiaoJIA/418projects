#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "image.h"
#include "CycleTimer.h"
#include "defines.h"

extern float toBW(int bytes, float sec);

__global__ void
chartest_kernel(char* distortions, int numDistortions, char* target, int tWidth, int tHeight, int numLocations, float* map) {
//    if(dID == 0)
//        printf("[T%d hello]\n", tid);

    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    
    int rangeW = tWidth - EDGE_DONT_BOTHER;

    int locId = blockId * NUM_THREADS_PER_BLOCK + threadId;
    if(locId >= numLocations)
        return;

    int tx_c = locId % rangeW;
    int ty_c = locId / rangeW;

    float* targetFix = (float*)target;

//    int totalLocs = w_range * h_range;

    float* dpixel = (float*) distortions;

    float maxVal = 0.0;
    for(int d = 0; d < numDistortions; d++) {
        // do all of the distortions at this location

        // calculate index of this distortion in buffer, pull out width/height
        Image * header = (Image*)dpixel;
        dpixel += (sizeof(Image) / sizeof(float));
        int dWidth = header->width;
        int dHeight = header->height;
        
        // calculate location to work on
        int tx_0 = tx_c - (dWidth / 2);
        int ty_0 = ty_c - (dHeight / 2);
        if(tx_0 < 0 || ty_0 < 0 || tx_0 + dWidth >= tWidth || ty_0 + dHeight >= tHeight)
            continue;

        float sum_let = 0.0;
        float sum_conv = 0.0;
        for(int dy = 0; dy < dHeight; dy++) {
            /* calculate index into target buffer */
            int ty = ty_0 + dy;
            int tIndex = ty * tWidth + tx_0;

            for(int dx = 0; dx < dWidth; dx++) {
            
                float t = targetFix[tIndex++];
                float d = *(dpixel++);

                /* Version 1
                 *   rewards matching as a percentage of pixels present */
#ifdef v1
                sum_let += d;
                sum_conv += (d * t);
#endif
                
                /* Version 2
                 *   rewards matching, punishes noise and missing */
#ifdef v2
                sum_let += d;
                //float match = d*t;
                //float noise = (1-d)*t;
                //float missing = d*(1-t);
                //sum_conv += (match - 0.3*noise - 0.7*missing);
                sum_conv += (2*d*t - 0.3*t - 0.7*d);
#endif

                /* Version 3
                 *   for recaptcha */
#ifdef v3
                sum_let += d;
                sum_conv += (2.0*d*t - 0.2*t - 0.3*d);
#endif
                
                /* Version 4
                 *   for recaptcha with horiz/vert edge tagging */
#ifdef v4
                float d_v = (d > 0.5)?1.0:0.0;
                float d_h = (d > 0.7 || (d > 0.3 && d < 0.5))?1.0:0.0;
                
                float t_v = (t > 0.5)?1.0:0.0;
                float t_h = (t > 0.7 || (t > 0.3 && t < 0.5))?1.0:0.0;

                sum_let += (d_v + d_h);
                sum_conv += 2.0*(d_v * t_v) + 2.0*(d_h * t_h);
                sum_conv -= 0.2*(t_h + t_v);
                sum_conv -= 0.3*(d_h + d_v);
#endif
                 
            }
        }
        //float val = (float)sum_conv;
        float val = (float)(sum_conv / sum_let);
        //float val = (float)(sum_conv / (sum_let + 40.0));
        //float val = (float)(sum_conv / max(sum_let, 110.0));
        val = (val < 0.0)?0.0:val;
        maxVal = (val > maxVal)?val:maxVal;
    }
    map[locId] = maxVal;

//    if(blockId == 0)
//        printf("[T%d found max of %f for (%d,%d) ]\n", threadId, maxVal, locX, locY);
    //__syncthreads();

}

/************************************
 * Reduce Columns Functions
 ***********************************/

__global__ void
reduce_columns_kernel(int rangeW, int rangeH, float* map, float* results) {
    int row = threadIdx.x;
    float max = 0.0;
    for(int c=0; c < rangeH; c++) {
        int mapIndex = (rangeW * c) + row;
        float thisV = map[mapIndex];
        max = (thisV>max)?thisV:max;
    }
    results[row] = max;
}

/************************************
 * CharTest Functions
 ***********************************/

double
charTest(char * distortionsBuf, int numDistortions, int totalDistortionBytes, char * device_target, int targetW, int targetH, int rangeW, int rangeH, float * resultBuf) {

    const int numLocations = rangeW * rangeH;

    // compute number of blocks and threads per block
    const int threadsPerBlock = NUM_THREADS_PER_BLOCK;
    const int blocks = ((numLocations + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK);

    const int mapBytes = numLocations * sizeof(float);
    const int resultBytes = rangeW * sizeof(float);

    // allocate letter buffers
    char * device_distortions;
    hipMalloc( &device_distortions, totalDistortionBytes );

    // allocate results buffers (full map, and final column-reduced)
    float * device_map;
    hipMalloc( &device_map, mapBytes );
    float * device_result;
    hipMalloc( &device_result, resultBytes );


    // copy letter buffers
    hipMemcpy(device_distortions, distortionsBuf, totalDistortionBytes, hipMemcpyHostToDevice);

    double kernelStartTime = CycleTimer::currentSeconds();
    // run map evaluation kernel
    chartest_kernel<<<blocks, threadsPerBlock>>>(device_distortions, numDistortions, device_target, targetW, targetH, numLocations, device_map);
    hipDeviceSynchronize();
    double kernelEndTime = CycleTimer::currentSeconds();

    // reduce columns
    reduce_columns_kernel<<<1, rangeW>>>(rangeW, rangeH, device_map, device_result);

    // copy result from GPU using hipMemcpy
    hipMemcpy( resultBuf, device_result, resultBytes, hipMemcpyDeviceToHost);
    
    //  free memory buffers on the GPU
    hipFree(device_distortions);
    hipFree(device_result);
    
    return( kernelEndTime - kernelStartTime);
}


/************************************
 * One time memory transfer
 ***********************************/

char * sendTarget(char* targetBuf, int targetBytes) {
    // allocate target buffer
    char* device_target;
    hipMalloc(&device_target, targetBytes);

    // copy target buffer
    hipMemcpy(device_target, targetBuf, targetBytes, hipMemcpyHostToDevice);

    return device_target;
}

void freeTarget(char* device_target) {
    hipFree(device_target);
}


void
printCudaInfo() {
    
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    printf("Found %d CUDA devices\n", deviceCount);
    
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}

/************************************
 ************************************
 ************************************
 * Sequential Functions for timing
 ************************************
 ************************************
 ***********************************/

void
chartest_kernel_sequential(float* distortions, int numDistortions, int maxDistortionSize, float* target, int tWidth, int tHeight, int numLocations, float* results, int blockId, int threadId) {
    int rangeW = tWidth - EDGE_DONT_BOTHER;

    int locId = blockId * NUM_THREADS_PER_BLOCK + threadId;
    if(locId >= numLocations)
        return;

    int tx_c = locId % rangeW;
    int ty_c = locId / rangeW;

//    int totalLocs = w_range * h_range;

    float maxVal = 0.0;
    for(int d = 0; d < numDistortions; d++) {
        // do all of the distortions at this location

        // calculate index of this distortion in buffer, pull out width/height
        int dIndex = (d * maxDistortionSize);
        int dWidth = int(distortions[dIndex++]);
        int dHeight = int(distortions[dIndex++]);
        
        // calculate location to work on
        int tx_0 = tx_c - (dWidth / 2);
        int ty_0 = ty_c - (dHeight / 2);
        if(tx_0 < 0 || ty_0 < 0 || tx_0 + dWidth >= tWidth || ty_0 + dHeight >= tHeight)
            continue;

        float sum_let = 0.0;
        float sum_conv = 0.0;
        for(int dy = 0; dy < dHeight; dy++) {
            for(int dx = 0; dx < dWidth; dx++) {
                /* calculate index into target buffer */
                int tx = tx_0 + dx;
                int ty = ty_0 + dy;
                int tIndex = ty * tWidth + tx;

                float t = target[tIndex];
                float d = distortions[dIndex++];

                /* Version 1
                 *   rewards matching as a percentage of pixels present */
#ifdef v1
                sum_let += d;
                sum_conv += (d * t);
#endif
                
                /* Version 2
                 *   rewards matching, punishes noise and missing */
#ifdef v2
                sum_let += d;
                //float match = d*t;
                //float noise = (1-d)*t;
                //float missing = d*(1-t);
                //sum_conv += (match - 0.3*noise - 0.7*missing);
                sum_conv += (2*d*t - 0.3*t - 0.7*d);
#endif

                /* Version 3
                 *   closeness of nearest pixel */
#ifdef v3
                sum_let += d;
                sum_conv += (2.0*d*t - 0.2*t - 0.3*d);
#endif
                
                /* Version 4
                 *   for recaptcha with horiz/vert edge tagging */
#ifdef v4
                float d_v = (d > 0.5)?1.0:0.0;
                float d_h = (d > 0.7 || (d > 0.3 && d < 0.5))?1.0:0.0;
                
                float t_v = (t > 0.5)?1.0:0.0;
                float t_h = (t > 0.7 || (t > 0.3 && t < 0.5))?1.0:0.0;

                sum_let += (d_v + d_h);
                sum_conv += 2.0*(d_v * t_v) + 2.0*(d_h * t_h);
                sum_conv -= 0.2*(t_h + t_v);
                sum_conv -= 0.3*(d_h + d_v);
#endif
                 
            }
        }
        //float val = (float)sum_conv;
        float val = (float)(sum_conv / sum_let);
        val = (val < 0.0)?0.0:val;
        maxVal = (val > maxVal)?maxVal:val;
    }
    results[locId] = maxVal;
}

void reduce_columns_sequential(int rangeW, int rangeH, int row, float* map, float* results) {
    float max = 0.0;
    for(int c=0; c < rangeH; c++) {
        int mapIndex = (rangeW * c) + row;
        float thisV = map[mapIndex];
        max = (thisV>max)?thisV:max;
    }
    results[row] = max;
}

double
charTestSequential(float * distortionsBuf, int numDistortions, int maxDistortionSize, float * targetBuf, int targetW, int targetH, int rangeW, int rangeH, float * resultBuf) {

    const int threadsPerBlock = NUM_THREADS_PER_BLOCK;
    int numLocations = rangeW * rangeH;
    const int blocks = ((numLocations + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK);
    
    const int mapBytes = numLocations * sizeof(float);
    float * mapBuf = (float*)malloc(mapBytes);

    double kernelStartTime = CycleTimer::currentSeconds();
    for(int b = 0; b < blocks; b++) {
        printf("Block %d/%d\n", b, blocks);
        for(int t = 0; t < threadsPerBlock; t++) {
            chartest_kernel_sequential(distortionsBuf, numDistortions, maxDistortionSize, targetBuf, targetW, targetH, numLocations, mapBuf, b, t);
        }
    }
    double kernelEndTime = CycleTimer::currentSeconds();

    for(int r = 0; r < rangeW; r++) {
        reduce_columns_sequential(rangeW, rangeH, r, mapBuf, resultBuf);
    }

    free(mapBuf);
    
    return( kernelEndTime - kernelStartTime);

}


