#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

#define TPB_X 16
#define TPB_Y 16
#define TPB (TPB_X * TPB_Y)

#define CIRC_LIST_START_SIZE 32

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    int numXRegions;
    int numYRegions;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

__device__ __inline__ int
circleInBox(
    float circleX, float circleY, float circleRadius,
    float boxL, float boxR, float boxT, float boxB)
{

    // clamp circle center to box (finds the closest point on the box)
    float closestX = (circleX > boxL) ? ((circleX < boxR) ? circleX : boxR) : boxL;
    float closestY = (circleY > boxB) ? ((circleY < boxT) ? circleY : boxT) : boxB;

    // is circle radius less than the distance to the closest point on
    // the box?
    float distX = closestX - circleX;
    float distY = closestY - circleY;

    if ( ((distX*distX) + (distY*distY)) <= (circleRadius*circleRadius) ) {
        return 1;
    } else {
        return 0;
    }
}


// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

//    if(!(blockIdx.x == 16 && blockIdx.y == 16))
//        return;

    int threadIndex = threadIdx.y * TPB_X + threadIdx.x;

    __shared__ int** circle_lists;
    __shared__ int* circle_list_counts;
    if(threadIndex == 0) {
        circle_lists = (int**) malloc(sizeof(int*) * TPB);
        if(circle_lists == NULL)
            printf("malloc failed\n");
        circle_list_counts = (int*) malloc(sizeof(int) * TPB);
        if(circle_list_counts == NULL)
            printf("malloc failed\n");
    }
    __syncthreads();
    
    int region_x = blockIdx.x; 
    int region_y = blockIdx.y;

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    //compute bounding box of region
    short region_xmin = TPB_X * region_x;
    short region_xmax = TPB_X * (region_x + 1) - 1;
    short region_ymin = TPB_Y * region_y;
    short region_ymax = TPB_Y * (region_y + 1) - 1;

    //account for boxes possibly pushed out of bounds by rounding
    region_xmax = (region_xmax < imageWidth) ? region_xmax : imageWidth - 1;
    region_ymax = (region_ymax < imageHeight) ? region_ymax : imageHeight - 1;

    // convert to normalized float coords
    float boxL = invWidth * region_xmin;
    float boxR = invWidth * region_xmax;
    float boxB = invHeight * region_ymin;
    float boxT = invHeight * region_ymax;

    /*************************************************
     * Phase 1
     *   build circle-list for each region
     *************************************************/

    int numCircles = cuConstRendererParams.numCircles;
    int circlesPerThread = numCircles / TPB;
    int circStart = threadIndex * circlesPerThread;
    int circEnd = circStart + circlesPerThread - 1;
    if(threadIndex == TPB - 1)
        circEnd = numCircles - 1;

    // malloc circle_list in device heap memory
    circle_list_counts[threadIndex] = 0;
    int circle_list_size = (CIRC_LIST_START_SIZE > circlesPerThread)? circlesPerThread: CIRC_LIST_START_SIZE;
    circle_lists[threadIndex] = (int*) malloc(sizeof(int) * circle_list_size);
    if(circle_lists[threadIndex] == NULL)
        printf("Malloc failed");

    //float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixel_x) + 0.5f), invHeight * (static_cast<float>(pixel_y) + 0.5f));
    
    for(int i = circStart; i <= circEnd; i++) {
          int index3 = 3 * i;

          // read position and radius
          float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
          float  rad = cuConstRendererParams.radius[i];

          if (circleInBox(p.x, p.y, rad, boxL, boxR, boxT, boxB)) {

           //add to this threads list of circles 
           if(circle_list_counts[threadIndex] == circle_list_size) {
              int* new_circle_list = (int*) malloc(2 * sizeof(int) * circle_list_size);
              if(new_circle_list == NULL) {
                printf("Malloc failed");
              }
              for(int j=0; j < circle_list_size; j++) {
                new_circle_list[j] = circle_lists[threadIndex][j];
              }
              free(circle_lists[threadIndex]);
              circle_lists[threadIndex] = new_circle_list;
              circle_list_size *= 2;
           }
           circle_lists[threadIndex][circle_list_counts[threadIndex]] = i;
           circle_list_counts[threadIndex] += 1;
         }
      }
      //if(blockIdx.x == 0 && blockIdx.y == 63)
        // printf("t-idx: %d circ-count: %d\n", threadIndex, circle_list_counts[threadIndex]);

    /*************************************************
     * Phase 2
     *    render each pixel in this region based off circle-list
     *************************************************/
    __syncthreads();
    //calculate my pixel coordinates
    int pixel_x = region_xmin + threadIdx.x;
    int pixel_y = region_ymin + threadIdx.y;

    //check that we're on screen - use region since's its already clamped
    if (!(pixel_x < imageWidth && pixel_y < imageHeight)) {
      //bail
      return;
    }

    int ci;

    for(int c = 0; c < TPB; c++) {

        for(int i = 0; i < circle_list_counts[c]; i++) {
            ci = circle_lists[c][i];
            int index3 = 3 * ci;

            // read position and radius
            float3 p = *(float3*)(&cuConstRendererParams.position[index3]);

            //get the pointer
            float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixel_y * imageWidth + pixel_x)]);
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixel_x) + 0.5f), invHeight * (static_cast<float>(pixel_y) + 0.5f));

            shadePixel(ci, pixelCenterNorm, p, imgPtr);
        }
    }

    __syncthreads();
    free(circle_lists[threadIndex]);
    __syncthreads();
    if (threadIndex == 0) {
        free(circle_lists);
        free(circle_list_counts);
    }
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    numXRegions = 0;
    numYRegions = 0;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.
    numXRegions = ((image->width-1) / TPB_X) + 1;   // rounding up
    numYRegions = ((image->height-1) / TPB_Y) + 1; // rounding up

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.numXRegions = numXRegions;
    params.numYRegions = numYRegions;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {

        // 256 threads per block is a healthy number
        dim3 blockDim(256, 1);
        dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
        hipDeviceSynchronize();
    }
}

void
CudaRenderer::render() {
    // 256 threads per block is a healthy number
    dim3 blockDim(numXRegions, numYRegions);
    dim3 gridDim( TPB_X, TPB_Y );

    //printf("launching kernels %dx%d blks %dx%d threads/blk", blockDim.x, gridDim);
    printf("image size %d x %d\n", image->width, image->height);
    printf("launching kernels (%dx%d b @ %dx%d tpb)\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);
    printf("num circles: %d\n", numCircles);
    
    kernelRenderCircles<<<blockDim, gridDim>>>();
    hipDeviceSynchronize();
}
